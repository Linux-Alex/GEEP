#include "GPUTree.h"
#include "../solutions/Solution.h"
#include "../nodes/Node.h"
#include "../nodes/FunctionNode.h"
#include "../nodes/TerminalNode.h"

void GPUTree::allocate(size_t max_nodes, size_t population_size) {
   capacity = max_nodes;
   population = population_size;

   hipMallocManaged(&nodes, max_nodes * population_size * sizeof(int));
   hipMallocManaged(&values, max_nodes * population_size * sizeof(float));
   hipMallocManaged(&children, max_nodes * 2 * population_size * sizeof(int));
   hipMallocManaged(&parent_indices, max_nodes * population_size * sizeof(int));
   hipMallocManaged(&node_counts, max_nodes * population_size * sizeof(size_t));
}

void GPUTree::free() {
   hipFree(nodes);
   hipFree(values);
   hipFree(children);
   hipFree(parent_indices);
   hipFree(node_counts);
}

__host__ void GPUTree::addSolution(int index, Solution *solution) {
   // TODO: Implementation to convert Solution to linear GPU format
}
